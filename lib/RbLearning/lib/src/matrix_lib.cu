
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <unistd.h>

__global__ void cuda_dot(double *m1, double *m2, int size_y, int size_v, int size_x, double *new_m) {
 	m2[(threadIdx.y * size_v) + threadIdx.x];
	double tmp = 0;
	for (int x = 0; x < size_x; ++x) {
		tmp += m1[(blockIdx.x * size_x) + x] * m2[(x * size_v) + threadIdx.x];
	}
	new_m[(blockIdx.x * size_v) + threadIdx.x] = tmp;
}

__global__ void cuda_mult(double *m1, double *m2, double *new_m) {
	new_m[blockIdx.x * blockDim.x + threadIdx.x] = m1[blockIdx.x * blockDim.x + threadIdx.x] * m2[blockIdx.x * blockDim.x + threadIdx.x];
}

__global__ void cuda_sub(double *m1, double *m2, double *new_m) {
	new_m[blockIdx.x * blockDim.x + threadIdx.x] = m1[blockIdx.x * blockDim.x + threadIdx.x] - m2[blockIdx.x * blockDim.x + threadIdx.x];
}

__global__ void cuda_add(double *m1, double *m2, double *new_m) {
	new_m[blockIdx.x * blockDim.x + threadIdx.x] = m1[blockIdx.x * blockDim.x + threadIdx.x] + m2[blockIdx.x * blockDim.x + threadIdx.x];
}

extern "C" {

	int find_nb_blocks(int size, int max_th) {
		int nb_block = 1;

		while (((float)size / (float)nb_block) > (float)max_th || (size % nb_block) != 0) {
			nb_block += 1;
		}

		return (nb_block);
	}

	double *dot(double *m1, double *m2, int size_y, int size_v, int size_x) {
		double *new_m 		= (double *)malloc((size_v * size_y) * sizeof(double));
		double *cuda_new_m 	= NULL;
		double *cuda_m1 	= NULL;
		double *cuda_m2 	= NULL;

		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if (error != 0) {
			fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error));
			exit(1);
		}

		hipMalloc((void**)&cuda_new_m, (size_y * size_v) * sizeof(double));
		memset(new_m, 0, (size_y * size_v) * sizeof(double));
		hipMalloc((void**)&cuda_m1, (size_x * size_y) * sizeof(double));
		hipMalloc((void**)&cuda_m2, (size_x * size_v) * sizeof(double));

		hipMemcpy(cuda_m1, m1, (size_x * size_y) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(cuda_m2, m2, (size_x * size_v) * sizeof(double), hipMemcpyHostToDevice);
		
		cuda_dot<<<size_y, size_v>>>(cuda_m1, cuda_m2, size_y, size_v, size_x, cuda_new_m);
		hipMemcpy(new_m, cuda_new_m, (size_y * size_v) * sizeof(double), hipMemcpyDeviceToHost);

		hipFree(cuda_new_m);
		hipFree(cuda_m1);
		hipFree(cuda_m2);
		return(new_m);
	}

	double *mult(double *m1, double *m2, int size) {
		double *new_m 		= (double *)malloc((size) * sizeof(double));
		double *cuda_new_m 	= NULL;
		double *cuda_m1 	= NULL;
		double *cuda_m2 	= NULL;

		dim3 numBlocks(find_nb_blocks(size, 1024));
		dim3 threadsPerBlock(size / numBlocks.x);

		hipMalloc((void**)&cuda_new_m, size * sizeof(double));
		memset(new_m, 0, size * sizeof(double));
		hipMalloc((void**)&cuda_m1, size * sizeof(double));
		hipMalloc((void**)&cuda_m2, size * sizeof(double));

		hipMemcpy(cuda_m1, m1, (size) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(cuda_m2, m2, (size) * sizeof(double), hipMemcpyHostToDevice);

		cuda_mult<<<numBlocks, threadsPerBlock>>>(cuda_m1, cuda_m2, cuda_new_m);
		hipMemcpy(new_m, cuda_new_m, size * sizeof(double), hipMemcpyDeviceToHost);

		hipFree(cuda_new_m);
		hipFree(cuda_m1);
		hipFree(cuda_m2);
		return(new_m);
	}

	double *transpose(double  *m, int size_y, int size_x)
	{
		double *new_m = (double *)malloc(sizeof(double) * (size_x * size_y));
		memset(new_m, 0, size_x * size_y * sizeof(double));

		for (int y = 0; y < size_y; ++y) {
			for (int x = 0; x < size_x; ++x) {
				new_m[x * size_y + y] = m[y * size_x + x];
			}
		}
		return (new_m);
	}

	double *substract(double *m1, double *m2, int size) {
		double *new_m 		= (double *)malloc((size) * sizeof(double));
		double *cuda_new_m 	= NULL;
		double *cuda_m1 	= NULL;
		double *cuda_m2 	= NULL;

		dim3 numBlocks(find_nb_blocks(size, 1024));
		dim3 threadsPerBlock(size / numBlocks.x);

		hipMalloc((void**)&cuda_new_m, size * sizeof(double));
		memset(new_m, 0, size * sizeof(double));
		hipMalloc((void**)&cuda_m1, size * sizeof(double));
		hipMalloc((void**)&cuda_m2, size * sizeof(double));

		hipMemcpy(cuda_m1, m1, (size) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(cuda_m2, m2, (size) * sizeof(double), hipMemcpyHostToDevice);
		
		cuda_sub<<<numBlocks, threadsPerBlock>>>(cuda_m1, cuda_m2, cuda_new_m);
		hipMemcpy(new_m, cuda_new_m, size * sizeof(double), hipMemcpyDeviceToHost);

		hipFree(cuda_new_m);
		hipFree(cuda_m1);
		hipFree(cuda_m2);
		return(new_m);
	}

	double *add(double *m1, double *m2, int size) {
		double *new_m 		= (double *)malloc((size) * sizeof(double));
		double *cuda_new_m 	= NULL;
		double *cuda_m1 	= NULL;
		double *cuda_m2 	= NULL;

    	dim3 numBlocks(find_nb_blocks(size, 1024));
		dim3 threadsPerBlock(size / numBlocks.x);

		hipMalloc((void**)&cuda_new_m, size * sizeof(double));
		memset(new_m, 0, size * sizeof(double));
		hipMalloc((void**)&cuda_m1, size * sizeof(double));
		hipMalloc((void**)&cuda_m2, size * sizeof(double));

		hipMemcpy(cuda_m1, m1, (size) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(cuda_m2, m2, (size) * sizeof(double), hipMemcpyHostToDevice);

		cuda_add<<<numBlocks, threadsPerBlock>>>(cuda_m1, cuda_m2, cuda_new_m);
		hipMemcpy(new_m, cuda_new_m, size * sizeof(double), hipMemcpyDeviceToHost);

		hipFree(cuda_new_m);
		hipFree(cuda_m1);
		hipFree(cuda_m2);
		return(new_m);
	}
}
