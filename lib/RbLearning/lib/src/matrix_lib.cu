#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include "matrix_lib.h"

__global__ void cuda_dot(const double *m1, const double *m2, int size_y, int size_v, int size_x, double *new_m) {
	double tmp = 0;
	for (int x = 0; x < size_x; ++x) {
		tmp += m1[(blockIdx.x * size_x) + x] * m2[(x * size_v) + threadIdx.x];
	}
	new_m[(blockIdx.x * size_v) + threadIdx.x] = tmp;
}

__global__ void cuda_mult(const double *m1, const double *m2, double *new_m) {
	new_m[blockIdx.x] = m1[blockIdx.x] * m2[blockIdx.x];
}

extern "C" {

	double *dot(const double *m1, const double *m2, int size_y, int size_v, int size_x) {
		double *new_m 		= (double *)malloc((size_v * size_y) * sizeof(double));
		double *cuda_new_m 	= NULL;
		double *cuda_m1 		= NULL;
		double *cuda_m2 		= NULL;

		hipMalloc((void**)&cuda_new_m, (size_y * size_v) * sizeof(double));
		hipMalloc((void**)&cuda_m1, (size_x * size_y) * sizeof(double));
		hipMalloc((void**)&cuda_m2, (size_x * size_v) * sizeof(double));

		hipMemcpy(cuda_m1, m1, (size_x * size_y) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(cuda_m2, m2, (size_x * size_v) * sizeof(double), hipMemcpyHostToDevice);

		cuda_dot<<<size_y, size_v>>>(cuda_m1, cuda_m2, size_y, size_v, size_x, cuda_new_m);
		hipMemcpy(new_m, cuda_new_m, (size_y * size_v) * sizeof(double), hipMemcpyDeviceToHost);

		hipFree(cuda_new_m);
		hipFree(cuda_m1);
		hipFree(cuda_m2);
		return(new_m);
	}

	double *mult(const double *m1, const double *m2, int size) {
		double *new_m 		= (double *)malloc((size) * sizeof(double));
		double *cuda_new_m 	= NULL;
		double *cuda_m1 		= NULL;
		double *cuda_m2 		= NULL;

		hipMalloc((void**)&cuda_new_m, size * sizeof(double));
		hipMalloc((void**)&cuda_m1, size * sizeof(double));
		hipMalloc((void**)&cuda_m2, size * sizeof(double));

		hipMemcpy(cuda_m1, m1, (size) * sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(cuda_m2, m2, (size) * sizeof(double), hipMemcpyHostToDevice);

		cuda_mult<<<size, 1>>>(cuda_m1, cuda_m2, cuda_new_m);
		hipMemcpy(new_m, cuda_new_m, size * sizeof(double), hipMemcpyDeviceToHost);

		hipFree(cuda_new_m);
		hipFree(cuda_m1);
		hipFree(cuda_m2);
		return(new_m);
	}

	double *transpose(const double  *m, int size_y, int size_x)
	{
		double *new_m = (double *)malloc(sizeof(double) * (size_x * size_y));

		for (int y = 0; y < size_y; ++y) {
			for (int x = 0; x < size_x; ++x) {
				new_m[x * size_y + y] = m[y * size_x + x];
			}
		}
		return (new_m);
	}

	double *subtract(const double *m1, const double *m2, int size_y, int size_x) {
		int size = size_x * size_y;
		double *new_m = (double *)malloc(sizeof(double) * size);

		for (int i = 0; i < size; ++i) {
			new_m[i] = m1[i] - m2[i];
		}
		return (new_m);
	}

	double *add(const double *m1, const double *m2, int size_y, int size_x) {
		int size = size_x * size_y;
		double *new_m = (double *)malloc(sizeof(double) * size);

		for (int i = 0; i < size; ++i) {
			new_m[i] = m1[i] + m2[i];
		}
		return (new_m);
	}
}
